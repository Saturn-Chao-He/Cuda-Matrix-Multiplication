#include "hip/hip_runtime.h"
//******************************************************************************
// Fall 2020
// Assignment: gpu.cu
// Instructor: Dr. Jiang
// Programmer: Chao He
//******************************************************************************
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <malloc.h>
#include <time.h>

// Function prototype
void initialData(int *, int *, int *, int);
hipError_t MatrixMalOnGPU(const int *, const int *, int *, const int, const int, const int);
__global__ void MatrixMalOnGPUKernel(const int *, const int *, int *, const int, const int, const int);

// Main function
int main(int argc, char **argv)
{
	printf("\n*****************************************************************************************\n");
    printf("\n%s Starting...\n", argv[0]);

    // Set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("\nUsing Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

	// result[M][S] = a[M][N] * b[N][S]
    int M = 1024, N = 1024, S = 1024;

	// Allocate the space of matrices A and B
	int * a = (int *)malloc(M * N * sizeof(int));
	if (NULL == a)
	{
		printf("the malloc of Matrix a is failed!\n");
		return 0;
    }
	int * b = (int *)malloc(N * S * sizeof(int));
	if (NULL == b)
	{
		printf("the malloc of Matrix b is failed!\n");
		return 0;
	}
	
	// Allocate the space of matrices 
	int *c = (int *)malloc(M * S * sizeof(int));
	if (NULL == c)
	{
		printf("the malloc of Matrix c is failed!\n");
		return 0;
    }

	// Initialize the matrix and print them 
	initialData(a, b, c, 1024*1024);

	//GPU excution
	hipError_t cudaStatus = MatrixMalOnGPU(a, b, c, M, N, S);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "MatrixMalOnGPU failed!");
		return 0;
    }
	
	// Print the result of C
	printf("\n");
	printf("\nThe result of matrix C:\n");
    for(int i = 0; i < 10; i++)  printf("%d ", c[i]);
    printf("\n");
	for(int i = 1024*1024-10; i < 1024*1024; i++)  printf("%d ", c[i]);  
    printf("\n");

    // free host memory
    free(a);
    free(b);
	free(c);
	printf("\n*****************************************************************************************\n");
    return 0;
}

// Initialize the matrix and print them 
void initialData(int *a, int *b, int *c, int size )
{
    // Fill 1 to matrix A and fill 2 to matrix B
    for (int i = 0; i < 1024*1024; i++)
    {
        a[i] = 1;
        b[i] = 2;
		c[i] = 0;
    }
    // Print A
    printf("\nThe matrix A: \n");
    for(int i = 0; i < 10; i++)  printf("%d ", a[i]);
    printf("\n");
	for(int i = 1024*1024-10; i < 1024*1024; i++)  printf("%d ", a[i]);  
    printf("\n");

    printf("\n");
	// Print B
    printf("\nThe matrix B: \n");
    for(int i = 0; i < 10; i++)  printf("%d ", b[i]);
    printf("\n");
	for(int i = 1024*1024-10; i < 1024*1024; i++)  printf("%d ", b[i]);  
    printf("\n");

	// Print C
	printf("\n");
    printf("\nThe matrix C: \n");
    for(int i = 0; i < 10; i++)  printf("%d ", c[i]);
    printf("\n");
	for(int i = 1024*1024-10; i < 1024*1024; i++)  printf("%d ", c[i]);  
    printf("\n");

    printf("\n");
}

// GPU excution
hipError_t MatrixMalOnGPU(const int *a, const int *b, int *result, const int M, const int N, const int S)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_result = 0;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	
	// Check the device
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!\n");		
	}

	// Malloc device global memory time
	hipEvent_t mallocStart, mallocFinish;
	float mallocElapsedTime;
	hipEventCreate(&mallocStart);
	hipEventCreate(&mallocFinish);
	hipEventRecord(mallocStart, 0);

	// Malloc device global memory 
	cudaStatus = hipMalloc((void **)&dev_a, M * N * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_a failed!\n");		
	}
	cudaStatus = hipMalloc((void **)&dev_b, N * S * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_b failed!\n");		
	}
	cudaStatus = hipMalloc((void **)&dev_result, M * S * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_result failed!\n");		
	}

	hipEventRecord(mallocFinish, 0);
	hipEventSynchronize(mallocFinish);
	hipEventElapsedTime(&mallocElapsedTime, mallocStart, mallocFinish);
	printf("\nThe time of memory allocating is %lf ms.\n", mallocElapsedTime);


	// transfer data time
	hipEvent_t transferStart, transferFinish;
	float transferElapsedTime;
	hipEventCreate(&transferStart);
	hipEventCreate(&transferFinish);
	hipEventRecord(transferStart, 0);
	// transfer data from host to device
	cudaStatus = hipMemcpy(dev_a, a, M * N * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudamemcpy dev_a failed!\n");		
	}
	cudaStatus = hipMemcpy(dev_b, b, N * S * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy dev_b failed!\n");		
	}

	hipEventRecord(transferFinish, 0);
	hipEventSynchronize(transferFinish);
	hipEventElapsedTime(&transferElapsedTime, transferStart, transferFinish);
	printf("\nThe time of data transfering is %lf ms.\n", transferElapsedTime);

	// GPU execution time
	hipEvent_t gpuStart, gpuFinish;
	float elapsedTime;
	hipEventCreate(&gpuStart);
	hipEventCreate(&gpuFinish);
	hipEventRecord(gpuStart, 0);

	// Maximum number of threads per block:           1024
	// Maximum sizes of each dimension of a block:    1024 x 1024 x 64
	// Maximum sizes of each dimension of a grid:     2147483647 x 65535 x 65535
	// Maximum number of threads per multiprocessor:  2048
	
	// matrix size
	int nx = 1024; // S
	int ny = 1024; // M
	// use 2D grid and 2D blocks
	int dimx = 16;
	int dimy = 16;
	dim3 block(dimx, dimy);
	dim3 grid( nx/block.x, ny/block.y );

	printf("\n");
	printf("Do matrix multiplication on GPU...");
    printf("\n");

	MatrixMalOnGPUKernel <<< grid, block >>> (dev_a, dev_b, dev_result, M, N, S);

	hipEventRecord(gpuFinish, 0);
	hipEventSynchronize(gpuFinish);
	hipEventElapsedTime(&elapsedTime, gpuStart, gpuFinish);
	printf("\nThe GPU excution time is %lf ms.\n", elapsedTime);

	// Check Kernel launch
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "MulKernel launch failed: %s!\n", hipGetErrorString(cudaStatus));
	}

	// Check synchronizaton
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize return Error code %d after Kernel launched!\n", cudaStatus);
	}

    // Copy kernel result back to host side
	cudaStatus = hipMemcpy(result, dev_result, M * S * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy result failed!\n");
    }
    
    // Free device global memory
    CHECK(hipFree(dev_a));
    CHECK(hipFree(dev_b));
    CHECK(hipFree(dev_result));

	return cudaStatus;
}

// Multiply the matrices on GPU
__global__ void MatrixMalOnGPUKernel(const int *a, const int *b, int *result, const int M, const int N, const int S)
{
	// global index: threadId = iy * nx + ix = iy * gridDim.x * blockDim.x + ix
	int threadId = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < M * S)
	{
		// linear global index, so /S to find the row and use %S to find the column
		int row = threadId / S; 
		int column = threadId % S;

		for (int i = 0; i < N; i++)
		{
			result[threadId] += a[row * N + i] * b[i * S + column];
		}
	}
}

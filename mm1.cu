#include "hip/hip_runtime.h"
//******************************************************************************
// Fall 2020
// Assignment: mm1.cu
// Instructor: Dr. Jiang
// Programmer: Chao He
//******************************************************************************
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <malloc.h>
#include <time.h>

// Function prototype
void initialData(int *, int *, int);
void MatrixMulOnCPU(const int *,const int *, int *,const int,const int,const int);
hipError_t MatrixMalOnGPU(const int *, const int *, int *, const int, const int, const int);
__global__ void MatrixMalOnGPUKernel(const int *, const int *, int *, const int, const int, const int);

// Main function
int main(int argc, char **argv)
{
    printf("\n%s Starting...\n", argv[0]);

    // Set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("\nUsing Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

	// result[M][S] = a[M][N] * b[N][S]
    int M = 16, N = 16, S = 16;

	// Allocate the space of matrices A and B
	int * a = (int *)malloc(M * N * sizeof(int));
	if (NULL == a)
	{
		printf("the malloc of Matrix a is failed!\n");
		return 0;
    }
	int * b = (int *)malloc(N * S * sizeof(int));
	if (NULL == b)
	{
		printf("the malloc of Matrix b is failed!\n");
		return 0;
    }
    
    // Allocate the space of matrices 
	//Store the results of CPU and GPU
	int * cpuResult = (int *)malloc(M * S * sizeof(int));
	if (NULL == cpuResult)
	{
		printf("the malloc of Matrix cpuResult is failed!\n");
		return 0;
    }
	int * gpuResult = (int *)malloc(M * S * sizeof(int));
	if (NULL == cpuResult)
	{
		printf("the malloc of Matrix gpuResult is failed!\n");
		return 0;
	}

	// Initialize the matrix and print them 
	initialData(a, b, 256 );

	// The time of CPU excution
	clock_t start, finish;
	double totalTime = 0.0;
	start = clock();

	//CPU excution
	MatrixMulOnCPU(a, b, cpuResult, M, N, S);
	finish = clock();
	totalTime = (double)(finish - start) / CLOCKS_PER_SEC;
	printf("\nThe CPU excution time is %lf seconds.\n", totalTime);

	//GPU excution
	hipError_t cudaStatus = MatrixMalOnGPU(a, b, gpuResult, M, N, S);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "MatrixMalOnGPU failed!");
		return 0;
    }
    
	// Print the result
	printf("\nThe result of CPU:\n");
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < S; j++)
		{
			printf("%d ", cpuResult[i * M + j]);
		}
		printf("\n");
	}
	printf("\nThe result of GPU:\n");
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < S; j++)
		{
			printf("%d ", gpuResult[i * M + j]);
		}
		printf("\n");
    }
    printf("\n");

	//Check the results of CPU and GPU
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < S; j++)
		{
			if (cpuResult[i * M + j] != gpuResult[i * M + j])
			{
				printf("the Results are not equal!\n");
				return 0;
			}
		}
	}

    // free host memory
    free(a);
    free(b);
    free(cpuResult);
    free(gpuResult);
    return 0;
}

// Initialize the matrix and print them 
void initialData(int *a, int *b, int size )
{
    // Fill 1 to matrix A and fill 2 to matrix B
    for (int i = 0; i < 256; i++)
    {
        a[i] = 1;
        b[i] = 2;
    }
    // Print A and B
    printf("\nThe matrix A: \n");
    for(int i = 0; i < 256; i++)
    {
        printf("%d ", a[i]);
        if( (i+1) % 16 == 0 )   
            printf("\n");
    }
    printf("\n");
    printf("\nThe matrix B: \n");
    for(int i = 0; i < 256; i++)
    {
        printf("%d ", b[i]);
        if( (i+1) % 16 == 0 )   
            printf("\n");
    }

    printf("\n");
}

// Matrix multiplication on CPU
void MatrixMulOnCPU(const int * a,const int * b, int *result,const int M,const int N,const int S)
{
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < S; j++)
		{
			int index = i * S + j;
			result[index] = 0;

			for (int k = 0; k < N; k++)
			{
				result[index] += a[i * N + k] * b[k * S + j];
			}
		}
	}
}

// GPU excution
hipError_t MatrixMalOnGPU(const int *a, const int *b, int *result, const int M, const int N, const int S)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_result = 0;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	
	// Check the device
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!\n");		
	}

	// Malloc device global memory 
	cudaStatus = hipMalloc((void **)&dev_a, M * N * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_a failed!\n");		
	}
	cudaStatus = hipMalloc((void **)&dev_b, N * S * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_b failed!\n");		
	}
	cudaStatus = hipMalloc((void **)&dev_result, M * S * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_result failed!\n");		
	}

	// transfer data from host to device
	cudaStatus = hipMemcpy(dev_a, a, M * N * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudamemcpy dev_a failed!\n");		
	}
	cudaStatus = hipMemcpy(dev_b, b, N * S * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy dev_b failed!\n");		
	}

	// Excution time
	hipEvent_t gpuStart, gpuFinish;
	float elapsedTime;
	hipEventCreate(&gpuStart);
	hipEventCreate(&gpuFinish);
	hipEventRecord(gpuStart, 0);

	// matrix size
	int nx = 16; // S
	int ny = 16; // M
	// use 2D grid and 2D blocks
	int dimx = 16;
	int dimy = 16;
	dim3 block(dimx, dimy);
	dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
	MatrixMalOnGPUKernel <<< grid, block >>> (dev_a, dev_b, dev_result, M, N, S);

	hipEventRecord(gpuFinish, 0);
	hipEventSynchronize(gpuFinish);
	hipEventElapsedTime(&elapsedTime, gpuStart, gpuFinish);
	printf("\nThe GPU excution time is %lf seconds.\n", elapsedTime / 1000.0);

	// Check Kernel launch
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "MulKernel launch failed: %s!\n", hipGetErrorString(cudaStatus));
	}

	// Check synchronizaton
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize return Error code %d after Kernel launched!\n", cudaStatus);
	}

    // Copy kernel result back to host side
	cudaStatus = hipMemcpy(result, dev_result, M * S * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy result failed!\n");
    }
    
    // Free device global memory
    CHECK(hipFree(dev_a));
    CHECK(hipFree(dev_b));
    CHECK(hipFree(dev_result));

	return cudaStatus;
}

// Multiply the matrices on GPU
__global__ void MatrixMalOnGPUKernel(const int *a, const int *b, int *result, const int M, const int N, const int S)
{
	// global index: threadId = iy * nx + ix = iy * gridDim.x * blockDim.x + ix
	int threadId = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < M * S)
	{
		// linear global index, so /S to find the row and use %S to find the column
		int row = threadId / S; 
		int column = threadId % S;

		result[threadId] = 0;
		for (int i = 0; i < N; i++)
		{
			result[threadId] += a[row * N + i] * b[i * S + column];
		}
	}
}
